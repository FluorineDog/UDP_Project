#include <time.h>
#include <cstring>
#include <fstream>
#include <iostream>
#include <random>
#include <sstream>
#include <string>

#include "hip/hip_runtime.h"


using namespace std;

#define PI 3.14159265358979323846
#define N 2048
#define M 256    //接收阵元到发射阵元的最大距离（阵元个数），所以接收孔径为2*M+1
#define ELE_NO 2048
#define OD 64
#define NSAMPLE 3750

int parallel_emit_sum = 1;

__device__ float dev_ele_coord_x[ELE_NO];
__device__ float dev_ele_coord_y[ELE_NO];    //写到纹理内存里面
__device__ float dev_filter_data[OD];        //filter parameter

float image_data[N * N] = {0};
int image_point_count[N * N] = {0};

// 滤波函数
// image_data as output location
// parallel_emit_sum is data trucks
// trans_sdata is raw input
// we have to use it against the best method
__global__ void calc_func(const int global_step, float *image_data,
                          int *point_count, const float *trans_sdata,
                          const int parallel_emit_sum) {
  int count = 1520;
  float fs = 25e6;
  float image_width = 200.0 / 1000;
  float image_length = 200.0 / 1000;
  float data_diameter = 220.0 / 1000;
  int point_length = data_diameter / count * fs + 0.5;
  float d_x = image_width / (N - 1);
  float d_z = image_length / (N - 1);

  int middot =
      -160;    //发射前1us开始接收，也就是约为12.5个点之后发射,数据显示约16个点

  int image_x_id = blockIdx.y;    //线
  int image_z_id = blockIdx.x;    //点
  int image_z_dim = gridDim.x;
  //blockIdx.x+blockIdx.y * gridDimx.x
  int recv_id = threadIdx.x;    //接收阵元

  __shared__ float cache_image[2 * M];
  __shared__ int cache_point[2 * M];
  int cacheIndex = threadIdx.x;

  if (image_x_id < N && image_z_id < N && recv_id < 2 * M) {
    float u = 0;
    int point_count_1 = 0;
    float z1 = -image_length / 2 + d_z * image_z_id;
    float x = -image_length / 2 + d_x * image_x_id;
    float xg = 0.0014;

    for (int step_offset = 0; step_offset < parallel_emit_sum; step_offset++) {
      int step = global_step + step_offset;
      int j = step - M + recv_id;    //接收阵元
      j = (j + ELE_NO) % ELE_NO;

      float disi =
          sqrtf((dev_ele_coord_x[step] - x) * (dev_ele_coord_x[step] - x) +
                (z1 - dev_ele_coord_y[step]) * (z1 - dev_ele_coord_y[step]));
      float disj = sqrtf((dev_ele_coord_x[j] - x) * (dev_ele_coord_x[j] - x) +
                         (z1 - dev_ele_coord_y[j]) * (z1 - dev_ele_coord_y[j]));
      float ilength = 112.0 / 1000;
      float imagelength = sqrtf(x * x + z1 * z1);
      float angle =
          acosf((ilength * ilength + disi * disi - imagelength * imagelength) /
                2 / ilength / disi);
      if ((disi >= 0.1 * 2 / 3 &&
           (abs(step - j) < 256 || abs(step - j) > 2048 - 256)) ||
          (disi >= 0.1 * 1 / 3 &&
           (abs(step - j) < 200 || abs(step - j) > 2048 - 200)) ||
          (disi >= 0 && (abs(step - j) < 100 || abs(step - j) > 2048 - 100))) {
        int num = (disi + disj) / count * fs + 0.5;

        if (((num + middot + (OD - 1 - 1) / 2) > 100) &&
            ((num + middot + (OD - 1 - 1) / 2) <= point_length) &&
            (angle < PI / 9)) {
          u += trans_sdata[(num + middot + (OD - 1 - 1) / 2) * ELE_NO + j +
                           step_offset * ELE_NO * NSAMPLE] *
               expf(xg * (num - 1));

          point_count_1 += 1;
        }
      }
    }
    cache_image[cacheIndex] = u;
    cache_point[cacheIndex] = point_count_1;

    __syncthreads();
    // sum up
    int step = blockDim.x / 2;
    while (step != 0) {
      if (cacheIndex < step) {
        cache_image[cacheIndex] += cache_image[cacheIndex + step];
        cache_point[cacheIndex] += cache_point[cacheIndex + step];
      }
      __syncthreads();
      step /= 2;
    }

    if (cacheIndex == 0) {
      int pixel_index = image_z_id + image_x_id * image_z_dim;    //线程块的索引
      image_data[pixel_index] = cache_image[0];
      point_count[pixel_index] = cache_point[0];
    }
  }
}

int main() {
  dim3 dim(1, 1);
  calc_func<<<dim, 512>>>(0, 0, 0, 0, 0);
}